#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)



//#define DEBUG

#ifdef  DEBUG
	#define print_debug(...) printf(__VA_ARGS__)
#else 
	#define print_debug(...) do {}while(0)
#endif //  

#define MIN(a,b) a>b ? b:a
#define CALC_Y(x,y,z) 0.299 * x + +0.587 * y + 0.114 * z

// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex_RGB_format;


__device__ double mask_put(int x, int y, int* maska)
{
	double brightness = 0;
	double grad = 0;
	int index_l_array[3] = {-1,0,1};
	int size_mask_x = 3;
	int size_mask_y = 3;
	uchar4 u_format;
	// int maska_X[9] = { -1,0,1,-2,0,2,-1,0,1};
	
	for(int i=0;i<size_mask_y * size_mask_x;i++)
	{
		u_format = tex2D(tex_RGB_format, x +index_l_array[i%3], y - index_l_array[i/3]);
		brightness = 0.299 * u_format.x + 0.587 * u_format.y + 0.114 * u_format.z;//CALC_Y(u_format.x, u_format.y, u_format.z);
		grad += brightness*(double)maska[8 - i];

	}
	return grad;
}

// __device__ double mask_y(int x, int y)
// {
// 	double brightness, grad;
// 	int index_l_array[3] = {-1,0,1};
// 	const int size_mask_x = 3;
// 	const int size_mask_y = 3;
// 	uchar4 u_format;
// 	int maska_Y[9] = { -1, -2, -1, 0,0,0,1,2,1}; 
	
// 	for(int i=0;i<size_mask_y * size_mask_x;i++)
// 	{
// 		u_format = tex2D(tex_RGB_format, x +index_l_array[i%size_mask_x], y - index_l_array[i/size_mask_y]);
// 		brightness = 0.299*u_format.x+0.587*u_format.y + 0.114*u_format.z;//CALC_Y(u_format.x, u_format.y, u_format.z);
// 		grad+=brightness*(double)maska_Y[(size_mask_x*size_mask_y-1) - i];

// 	}
// 	return grad;
// }


__global__ void kernel(uchar4 *out, int height,int widht, int* maska_X, int* maska_Y)
{
	int indx = blockDim.x * blockIdx.x + threadIdx.x;
	int indy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	
	int x,y;

    uchar4 p_global;	

	// const int size_mask_x = 3;
	// const int size_mask_y = 3;
	

	
 	
	
	// double grad;
	// double temp_G;

	for (y = indy; y < height; y += offsety)
	{
		
		for (x = indx; x < widht; x += offsetx)
		{
            p_global = tex2D(tex_RGB_format, x,y);

			double gx = mask_put(x,y, maska_X);
			double gy = mask_put(x,y, maska_Y);
			
			double grad = sqrtf(gx*gx + gy*gy);
			// print_debug("gx = %d, gy = %d, grad = %d\n", gx,gy,grad);
			if(grad>UCHAR_MAX) { 
				// print_debug("max\n");
				grad = UCHAR_MAX;}
			out[y * widht + x] = make_uchar4(grad, grad, grad, p_global.w);
			// print_debug("out[%d] = %d\n",y * widht + x, out[y * widht + x]);


		}
	}
	
}

int main()
{

	
	int widht, height;
	char name_in[255];
    char name_out[255];
	scanf("%s",name_in);
    scanf("%s",name_out);

	FILE* file_read = fopen(name_in, "rb");
	
	if (NULL == file_read)
	{
		fprintf(stderr, "File read - error <do not found file>\n");
		return 0;
	}
	int size_mask = 9;
	int maska_X[9] = { -1,0,1,-2,0,2,-1,0,1};
	int maska_Y[9] = { -1, -2, -1, 0,0,0,1,2,1}; 
	
	
	fread(&widht, sizeof(int), 1, file_read);
	// print_debug("height = %d\n", widht);

	fread(&height, sizeof(int), 1, file_read);
	
	// print_debug("widht = %d\n", height);
	
	uchar4* image = (uchar4*)malloc(sizeof(uchar4) * widht* height);
	

	
	
	fread(image, sizeof(uchar4), widht * height, file_read);
	
		
	
	
	
	

	
	fclose(file_read);

	hipArray *cuda_image;

	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();


	
	int *dev_maska_X;
	int *dev_maska_Y;

	CSC(hipMalloc(&dev_maska_X, sizeof(int) *size_mask));
	CSC(hipMalloc(&dev_maska_Y, sizeof(int) *size_mask));

	CSC(hipMemcpy(dev_maska_X, maska_X, size_mask*sizeof(int),hipMemcpyHostToDevice ));
	CSC(hipMemcpy(dev_maska_Y, maska_Y, size_mask*sizeof(int),hipMemcpyHostToDevice ));

	CSC(hipMallocArray(&cuda_image, &ch, widht,height));

	CSC(hipMemcpyToArray(cuda_image, 0, 0, image, sizeof(uchar4) * widht * height, hipMemcpyHostToDevice));

	
	tex_RGB_format.addressMode[0] = hipAddressModeClamp;	
	tex_RGB_format.addressMode[1] = hipAddressModeClamp;
	tex_RGB_format.channelDesc    = ch;
	tex_RGB_format.filterMode     = hipFilterModePoint;	
	tex_RGB_format.normalized     = false;

	CSC(hipBindTextureToArray(tex_RGB_format, cuda_image, ch));


	uchar4* cuda_image_out;
	
	CSC(hipMalloc(&cuda_image_out, sizeof(uchar4) * widht * height));

	kernel << < dim3(32, 32), dim3(32, 32) >> > (cuda_image_out,height,widht, dev_maska_X, dev_maska_Y);

	CSC(hipGetLastError());

	CSC(hipMemcpy(image, cuda_image_out, sizeof(uchar4) * widht * height, hipMemcpyDeviceToHost));

	CSC(hipUnbindTexture(tex_RGB_format));




	file_read = fopen(name_out, "wb");

	if (NULL == file_read)
	{
		fprintf(stderr, "File  write - error <do not found file>\n");
		return 0;
	}


	fwrite(&widht, sizeof(int), 1, file_read);
	fwrite(&height, sizeof(int), 1, file_read);
	fwrite(image, sizeof(uchar4), widht * height, file_read);
	fclose(file_read);


	CSC(hipFreeArray(cuda_image));
	CSC(hipFree(cuda_image_out));
	CSC(hipFree(dev_maska_X));
	CSC(hipFree(dev_maska_Y));
	free(image);
	return 0;
}
